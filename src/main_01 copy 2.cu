#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int index = threadIdx.x;
    int stribe = blockDim.x;

    for (int i = index; i < n; i += stribe) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate memory on CPU
    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate memory on GPU
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Copy data from CPU to GPU
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    //vector_add<<<1, 1>>>(d_out, d_a, d_b, N);
    vector_add<<<1, 256>>>(d_out, d_a, d_b, N);

    for (int i = 0; i < N; i++) {
        assert(fabs(out[i] - a[i] - b[i] < MAX_ERR));
    }

    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}