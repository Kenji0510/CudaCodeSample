#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


//#define N 10000000
#define N 100000000
#define MAX_ERR 1e-6

__global__ void vector_subtract(uint8_t *out, uint8_t *a, uint8_t *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        int temp = a[tid] - b[tid];
        out[tid] = (temp < 0) ? 0 : temp;
    }
}

int main() {
    uint8_t *a, *b, *out;
    uint8_t *d_a, *d_b, *d_out;

    // Allocate memory on CPU
    a = (uint8_t*)malloc(sizeof(uint8_t) * N);
    b = (uint8_t*)malloc(sizeof(uint8_t) * N);
    out = (uint8_t*)malloc(sizeof(uint8_t) * N);

    for (int i = 0; i < N; i++) {
        a[i] = 255;
        b[i] = 20;
    }

    // Allocate memory on GPU
    hipMalloc((void**)&d_a, sizeof(uint8_t) * N);
    hipMalloc((void**)&d_b, sizeof(uint8_t) * N);
    hipMalloc((void**)&d_out, sizeof(uint8_t) * N);

    // Copy data from CPU to GPU
    hipMemcpy(d_a, a, sizeof(uint8_t) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(uint8_t) * N, hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = ((N + block_size) / block_size);
    vector_subtract<<<grid_size, block_size>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(uint8_t) * N, hipMemcpyDeviceToHost);

    printf("out[0] = %d\n", out[1]);
    printf("PASSED\n");

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}